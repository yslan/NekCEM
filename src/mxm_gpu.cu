#include <stdio.h>
#include <hip/hip_runtime.h>


#define KERNEL 1
#define TILE 16

extern "C" {
  void local_grad3_gpu_(double* u1r, double* u1s, double* u1t,  
                        double* u2r, double* u2s, double* u2t,  
                        double* u3r, double* u3s, double* u3t,  
                        double* u1 , double* u2 , double* u3 ,  
                        double* dxm1,   int* n,      int* nelts);
  void mxm_std_gpu_(double* a, int* m, double* b, int* n, double* c, int* p);
}


void print_array(double* a, int m, int n){
  int i,j,k=0;
  for(j=0; j<n; j++){
    for(i=0; i<m; i++){
      printf("array[%d][%d]=%E\n",i,j,a[k++]);
    }
  }
}
__global__ void mxm_vanilla(double* a, const int m, double* b, const int n, double* c, const int p
                           ,const int nelts, const int ldims){
  const int row=blockIdx.y*blockDim.y+threadIdx.y;
  const int col=blockIdx.x*blockDim.x+threadIdx.x;
  if(row<m && col<p){//eliminate out-of-bounds threads
    int lda=(ldims&0x1)*m*n //if a's bit (0x1) is set, its leading dim is of size m*n 
      , ldb=(ldims&0x2)*n*p
      , ldc=(ldims&0x4)*m*p;
    for(int e=0; e<nelts; e++){ // might need to launch 1 thread per element
      double s=0.0;
      for(int k=0; k<n; k++){
        s+=a[e*lda+k*m+row]*b[e*ldb+col*n+k];
      }
      c[e*ldc+col*m+row]=s;
    }
  }
}
__global__ void mxm_1d(double* a, const int m, double* b, const int n, double* c, const int p){
  const int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i<m){
    for(int k=0; k<p; k++){
      double s=0.0;
      for(int j=0; j<n; j++){
        s+=a[j*m+i]*b[k*n+j];
      }
      c[k*m+i]=s;
    }
  }
}
__global__ void mxm_shared(double* a, const int m, double* b, const int n, double* c, const int p){
  __shared__ double as[TILE][TILE];
  __shared__ double bs[TILE][TILE];
  int bx=blockIdx.x, by=blockIdx.y, tx=threadIdx.x, ty=threadIdx.y;
  const int row=by*TILE+ty;
  const int col=bx*TILE+tx;
  double s=0.0;
  for(int t=0;t<m/TILE;t++){
    as[ty][tx]=a[col*m+t*TILE+tx];
    bs[ty][tx]=b[col*n+t*TILE+ty];
    __syncthreads();
    for(int k=0; k<TILE; k++){
      s+=as[ty][k]*bs[k][tx];
    }
    __syncthreads();
    c[col*m+row]=s;
  }
}
void mxm_std_gpu_(double* a, int* m, double* b, int* n, double* c, int* p){
  //printf("mxm_gpu: m=%d,n=%d,p=%d\n",*m,*n,*p);
  //print_array(c,*m,*p);
  /*device variables*/
  double *dev_a, *dev_b, *dev_c;
  int sizeofA=*m*(*n)*sizeof(double)
    , sizeofB=*n*(*p)*sizeof(double)
    , sizeofC=*m*(*p)*sizeof(double);
  /*malloc and memcopy data from host to device*/
  hipMalloc(&dev_a,sizeofA);
  hipMalloc(&dev_b,sizeofB);
  hipMalloc(&dev_c,sizeofC);
  hipMemcpy(dev_a,a,sizeofA,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,sizeofB,hipMemcpyHostToDevice);
  /*thread dimensions*/
  dim3 dimBlock, dimGrid;
#if KERNEL==1
  dimBlock.x=TILE; dimGrid.x=(*p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(*m+dimBlock.y-1)/dimBlock.y;
  mxm_vanilla<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p,1,0);
#elif KERNEL==2
  dimBlock.x=TILE; dimGrid.x=(*m+dimBlock.x-1)/dimBlock.x;
  mxm_1d<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p);
#else
  dimBlock.x=TILE; dimGrid.x=(*p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(*m+dimBlock.y-1)/dimBlock.y;
  mxm_shared<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p);
#endif
  //printf("mxm_gpu: dimGrid.x=%d,dimGrid.y=%d\n",dimGrid.x,dimGrid.y);
  /*memcopy from device to host*/
  hipMemcpy(c,dev_c,sizeofC,hipMemcpyDeviceToHost);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  hipDeviceSynchronize();
}
void mxm_gpu2(double* a, int as, int m
             ,double* b, int bs, int n
             ,double* c, int cs, int p
             ,int nelts, int mask){
  //printf("mxm_gpu: m=%d,n=%d,p=%d\n",*m,*n,*p);
  //print_array(c,*m,*p);
  /*device variables*/
  double *dev_a, *dev_b, *dev_c;
  int sizeofA=as*sizeof(double)
    , sizeofB=bs*sizeof(double)
    , sizeofC=cs*sizeof(double);
  /*malloc and memcopy data from host to device*/
  hipMalloc(&dev_a,sizeofA);
  hipMalloc(&dev_b,sizeofB);
  hipMalloc(&dev_c,sizeofC);
  hipMemcpy(dev_a,a,sizeofA,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,sizeofB,hipMemcpyHostToDevice);
  /*thread dimensions*/
  dim3 dimBlock, dimGrid;
#if KERNEL==1
  dimBlock.x=TILE; dimGrid.x=(p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(m+dimBlock.y-1)/dimBlock.y;
  mxm_vanilla<<<dimGrid,dimBlock>>>(dev_a,m, dev_b,n, dev_c,p, nelts,mask);
#elif KERNEL==2
  dimBlock.x=TILE; dimGrid.x=(m+dimBlock.x-1)/dimBlock.x;
  mxm_1d<<<dimGrid,dimBlock>>>(dev_a,m,dev_b,n,dev_c,p);
#else
  dimBlock.x=TILE; dimGrid.x=(p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(m+dimBlock.y-1)/dimBlock.y;
  mxm_shared<<<dimGrid,dimBlock>>>(dev_a,m,dev_b,n,dev_c,p);
#endif
  //printf("mxm_gpu: dimGrid.x=%d,dimGrid.y=%d\n",dimGrid.x,dimGrid.y);
  /*memcopy from device to host*/
  hipMemcpy(c,dev_c,sizeofC,hipMemcpyDeviceToHost);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  hipDeviceSynchronize();
}
void local_grad3_gpu_(double* u1r, double* u1s, double* u1t,  
                      double* u2r, double* u2s, double* u2t,  
                      double* u3r, double* u3s, double* u3t,  
                      double* u1 , double* u2 , double* u3 ,  
                      double* dxm1,   int* n  ,    int* nelts){
  // foreach e in 0..nelts
  //   u*r_{NxN^2} = d_{NxN} * u*_{NxN^2}^{e} // * is either 1, 2 or 3
  //   foreach k in 0..N
  //     u*s_{NxN}^{k} = u*_{NxN}^{k,e} * dt_{NxN}
  //   u*t_{N^2xN} = u*_{N^2xN}^{e} * dt_{NxN}
//  int n2=*n*(*n)
//    , n3=*n*n2;
//    , npts=n3*(*nelts);
//  // calc u1r
//  mxm_gpu2(dxm1,n2  ,*n
//          ,u1  ,npts,*n
//          ,u1r ,npts,n2
//          ,*nelts,2);
}

