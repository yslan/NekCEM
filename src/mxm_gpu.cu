/*
 * mxm_gpu.cu
 *  @author azamat, mmin
 *  @since  July 13, 2012
 */

#include <stdio.h>
#include <hip/hip_runtime.h>


#define KERNEL  1
#define TILE   16 //autotune-able

extern "C" {
  void mxm_std_gpu_(double* a, int* m, double* b, int* n, double* c, int* p);
  void local_grad3_gpu_(
    double* u1r, double* u1s, double* u1t,
    double* u2r, double* u2s, double* u2t,
    double* u3r, double* u3s, double* u3t,
    double* u1 , double* u2 , double* u3 ,
    double* dxm, double* dxtm, int* n, int* nelts, int* rank);
  void curl_gpu_(
    double* u1r, double* u1s, double* u1t,
    double* u2r, double* u2s, double* u2t,
    double* u3r, double* u3s, double* u3t,
    double* rxmn,double* sxmn,double* txmn,
    double* rymn,double* symn,double* tymn,
    double* rzmn,double* szmn,double* tzmn,
    double* w1,  double* w2,  double* w3, double* w3m, int* nxyz, int* nelts);
}

struct memptr {
  int sync;
  int sz;
  double* host;
  double* dev;
};
typedef struct memptr memptr_t;

// basic curl kernel impl
__global__ void curl_vanilla(
    double* rxmn,double* rymn,double* rzmn,
    double* sxmn,double* symn,double* szmn,
    double* txmn,double* tymn,double* tzmn,
    double* u1r, double* u1s, double* u1t,
    double* u2r, double* u2s, double* u2t,
    double* u3r, double* u3s, double* u3t,
    double* w3m, const int nxyz, const int nelts,
    double* w1,  double* w2,  double* w3){
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  double w3mk;
  int k=0;
  for(int e=0; e<nelts; e++){
    k=e*nxyz+tid;
    w3mk=w3m[k];

    w1[k]= w3mk*u3r[k]*rymn[k]
         + w3mk*u3s[k]*symn[k]
         + w3mk*u3t[k]*tymn[k]
         - w3mk*u2r[k]*rzmn[k]
         - w3mk*u2s[k]*szmn[k]
         - w3mk*u2t[k]*tzmn[k];

    w2[k]= w3mk*u1r[k]*rzmn[k]
         + w3mk*u1s[k]*szmn[k]
         + w3mk*u1t[k]*tzmn[k]
         - w3mk*u3r[k]*rxmn[k]
         - w3mk*u3s[k]*sxmn[k]
         - w3mk*u3t[k]*txmn[k];

    w3[k]= w3mk*u2r[k]*rxmn[k]
         + w3mk*u2s[k]*sxmn[k]
         + w3mk*u2t[k]*txmn[k]
         - w3mk*u1r[k]*rymn[k]
         - w3mk*u1s[k]*symn[k]
         - w3mk*u1t[k]*tymn[k];
  }
}

// basic multi-mxm impl
__global__ void mxm_vanilla(double* a, const int m, double* b, const int n, double* c, const int p
                           ,const int nelts, const int ldims){
  const int row=blockIdx.y*blockDim.y+threadIdx.y;
  const int col=blockIdx.x*blockDim.x+threadIdx.x;
  double s;
  if(row<m && col<p){ //eliminate out-of-bounds threads
    int lda=(ldims&0x1)*m*n //if a's bit (0x1) is set, its leading dim is of size m*n 
      , ldb=((ldims&0x2)>>1)*n*p
      , ldc=((ldims&0x4)>>2)*m*p
      , ldai=((ldims&0x8)>>3)*m*n //for a's inner dimension
      , ldci=((ldims&0x8)>>3)*m*p;
    //printf("row=%d,col=%d,m=%d,n=%d,p=%d,nelts=%d,ldims=%d,lda=%d,ldb=%d,ldc=%d,ldai=%d,ldci=%d\n",row,col,m,n,p,nelts,ldims,lda,ldb,ldc,ldai,ldci);
    if(ldims<8){ //no inner iterations
      for(int e=0; e<nelts; e++){ // might need to launch 1 thread per element
        s=0.0;
        for(int k=0; k<n; k++){
          s+=a[e*lda+k*m+row]*b[e*ldb+col*n+k];
        }
        c[e*ldc+col*m+row]=s;
      }
    }else{
      for(int e=0; e<nelts; e++){ // might need to launch 1 thread per element
        for(int i=0; i<m; i++){
          s=0.0;
          for(int k=0; k<n; k++){
            s+=a[e*lda+i*ldai+k*m+row]*b[e*ldb+col*n+k];
          }
          c[e*ldc+i*ldci+col*m+row]=s;
        }
      }
    }
  }
}


// mxm with 1D arrays
__global__ void mxm_1d(double* a, const int m, double* b, const int n, double* c, const int p){
  const int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i<m){
    for(int k=0; k<p; k++){
      double s=0.0;
      for(int j=0; j<n; j++){
        s+=a[j*m+i]*b[k*n+j];
      }
      c[k*m+i]=s;
    }
  }
}


// mxm with 2D arrays
__global__ void mxm_shared(double* a, const int m, double* b, const int n, double* c, const int p){
  __shared__ double as[TILE][TILE];
  __shared__ double bs[TILE][TILE];
  int bx=blockIdx.x, by=blockIdx.y, tx=threadIdx.x, ty=threadIdx.y;
  const int row=by*TILE+ty;
  const int col=bx*TILE+tx;
  double s=0.0;
  for(int t=0;t<m/TILE;t++){
    as[ty][tx]=a[col*m+t*TILE+tx];
    bs[ty][tx]=b[col*n+t*TILE+ty];
    __syncthreads();
    for(int k=0; k<TILE; k++){
      s+=as[ty][k]*bs[k][tx];
    }
    __syncthreads();
    c[col*m+row]=s;
  }
}


// globally-visible basic mxm implementation for small matrices
void mxm_std_gpu_(double* a, int* m, double* b, int* n, double* c, int* p){
  /*device variables*/
  double *dev_a, *dev_b, *dev_c;
  int sizeofA=*m*(*n)*sizeof(double)
    , sizeofB=*n*(*p)*sizeof(double)
    , sizeofC=*m*(*p)*sizeof(double);
  /*malloc and memcopy data H2D*/
  hipMalloc(&dev_a,sizeofA);
  hipMalloc(&dev_b,sizeofB);
  hipMalloc(&dev_c,sizeofC);
  hipMemcpy(dev_a,a,sizeofA,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,sizeofB,hipMemcpyHostToDevice);
  /*thread dimensions*/
  dim3 dimBlock, dimGrid;
#if KERNEL==1
  dimBlock.x=TILE; dimGrid.x=(*p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(*m+dimBlock.y-1)/dimBlock.y;
  mxm_vanilla<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p,1,0);
#elif KERNEL==2
  dimBlock.x=TILE; dimGrid.x=(*m+dimBlock.x-1)/dimBlock.x;
  mxm_1d<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p);
#else
  dimBlock.x=TILE; dimGrid.x=(*p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(*m+dimBlock.y-1)/dimBlock.y;
  mxm_shared<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p);
#endif
  /*memcopy D2H*/
  hipMemcpy(c,dev_c,sizeofC,hipMemcpyDeviceToHost);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  hipDeviceSynchronize();
}


// sets up the aggregated mxm kernel launch
void mxm_gpu2(double* a, int as, int m
             ,double* b, int bs, int n
             ,double* c, int cs, int p
             ,int nelts, int mask, int dev){
  hipSetDevice(dev);
  /*device variables*/
  double *dev_a, *dev_b, *dev_c;
  int sizeofA=as*sizeof(double)
    , sizeofB=bs*sizeof(double)
    , sizeofC=cs*sizeof(double);
  /*malloc and memcopy H2D*/
  hipMalloc(&dev_a,sizeofA);
  hipMalloc(&dev_b,sizeofB);
  hipMalloc(&dev_c,sizeofC);
  hipMemcpy(dev_a,a,sizeofA,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,sizeofB,hipMemcpyHostToDevice);
  /*thread grid dimensions*/
  dim3 dimBlock, dimGrid;
  dimBlock.x=TILE; dimGrid.x=(p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(m+dimBlock.y-1)/dimBlock.y;
  mxm_vanilla<<<dimGrid,dimBlock>>>(dev_a,m, dev_b,n, dev_c,p, nelts,mask);
  /*memcopy D2H*/
  hipMemcpy(c,dev_c,sizeofC,hipMemcpyDeviceToHost);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  hipDeviceSynchronize();
}

// sets up the aggregated mxm kernel launch
void mxm_gpu_agg(memptr_t *a, int m
                ,memptr_t *b, int n
                ,memptr_t *c, int p
                ,int nelts, int mask, int dev){
  hipSetDevice(dev);
  /*malloc and memcopy H2D*/
  if (a->sync&0x1) { // if need to malloc
    hipMalloc(&a->dev,a->sz);
    a->sync^=0x1;
  }
  if (b->sync&0x1) {
    hipMalloc(&b->dev,b->sz);
    b->sync^=0x1;
  }
  if (c->sync&0x1) {
    hipMalloc(&c->dev,c->sz);
    c->sync^=0x1;
  }
  if (a->sync&0x2) { // if need to memcpy H2D
    hipMemcpy(a->dev,a->host,a->sz,hipMemcpyHostToDevice);
    a->sync^=0x2;
  }
  if (b->sync&0x2) {
    hipMemcpy(b->dev,b->host,b->sz,hipMemcpyHostToDevice);
    b->sync^=0x2;
  }
  if (c->sync&0x2) {
    hipMemcpy(c->dev,c->host,c->sz,hipMemcpyHostToDevice);
    c->sync^=0x2;
  }
  /*thread grid dimensions*/
  dim3 dimBlock, dimGrid;
  dimBlock.x=TILE; dimGrid.x=(p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(m+dimBlock.y-1)/dimBlock.y;
  mxm_vanilla<<<dimGrid,dimBlock>>>(a->dev,m, b->dev,n, c->dev,p, nelts,mask);
  /*memcopy D2H and dealloc*/
  if (a->sync&0x4) { // if need to memcpy D2H
    hipMemcpy(a->host,a->dev,a->sz,hipMemcpyDeviceToHost);
    a->sync^=0x4;
  }
  if (b->sync&0x4) {
    hipMemcpy(b->host,b->dev,b->sz,hipMemcpyDeviceToHost);
    b->sync^=0x4;
  }
  if (c->sync&0x4) {
    hipMemcpy(c->host,c->dev,c->sz,hipMemcpyDeviceToHost);
    c->sync^=0x4;
  }
  if (a->sync&0x8) { // if need to dealloc
    hipFree(a->dev);
    a->sync^=0x8;
  }
  if (b->sync&0x8) {
    hipFree(b->dev);
    b->sync^=0x8;
  }
  if (c->sync&0x8) {
    hipFree(c->dev);
    c->sync^=0x8;
  }
}


/**
 * Performs aggregated mxm for all elements at once.
 *
 * foreach e in 0..nelts
 *   u@r_{NxN^2} = d_{NxN} * u@_{NxN^2}^{e} // here @ is either 1, 2 or 3
 *   foreach k in 0..N
 *     u@s_{NxN}^{k} = u@_{NxN}^{k,e} * dt_{NxN}
 *   u@t_{N^2xN} = u@_{N^2xN}^{e} * dt_{NxN}
 */
void local_grad3_gpu_(double* u1r, double* u1s, double* u1t,  
                      double* u2r, double* u2s, double* u2t,  
                      double* u3r, double* u3s, double* u3t,  
                      double* u1 , double* u2 , double* u3 ,  
                      double* d  , double* dt , int* n, int* nelts, int* rank){
  int n2=*n*(*n), n3=*n*n2, npts=n3*(*nelts);
  //double *dd  =NULL, *ddt =NULL;
  //double *du1 =NULL, *du2 =NULL, *du3 =NULL;
  //double *du1r=NULL, *du2r=NULL, *du3r=NULL;
  // sync flags: 0x1->allocate, 0x2->copy H2D, 0x4->copy D2H, 0x8->deallocate
  //memptr_t ud   = {0x0011,sizeof(double)*n2, d ,dd};
  //memptr_t udt  = {0x0011,sizeof(double)*n2, dt,ddt};
  //memptr_t uu1  = {0x0011,sizeof(double)*npts, u1 ,du1 };
  //memptr_t uu2  = {0x0011,sizeof(double)*npts, u2 ,du2 };
  //memptr_t uu3  = {0x0011,sizeof(double)*npts, u3 ,du3 };
  //memptr_t uu1r = {0x1101,sizeof(double)*npts, u1r,du1r};
  //memptr_t uu2r = {0x1101,sizeof(double)*npts, u2r,du2r};
  //memptr_t uu3r = {0x1101,sizeof(double)*npts, u3r,du3r};

  int devs = 0;
  hipGetDeviceCount(&devs);
  int devid = *rank%2;

  if (devs==1) {
    //       d_{NxN}   *  u*_{NxN^2} = u*r_{NxN^2}   foreach e
    mxm_gpu2(d,n2,*n,     u1,npts,*n,  u1r,npts,n2,  *nelts,6, 0);
    mxm_gpu2(d,n2,*n,     u2,npts,*n,  u2r,npts,n2,  *nelts,6, 0);
    mxm_gpu2(d,n2,*n,     u3,npts,*n,  u3r,npts,n2,  *nelts,6, 0);
    //mxm_gpu_agg(&ud,*n,    &uu1,*n,  &uu1r,n2,  *nelts,6, 0);
    //mxm_gpu_agg(&ud,*n,    &uu2,*n,  &uu2r,n2,  *nelts,6, 0);
    //ud.sync=0x1000;
    //mxm_gpu_agg(&ud,*n,    &uu3,*n,  &uu3r,n2,  *nelts,6, 0);
  
    //       u*_{NxN}  *  dt_{NxN}  =  u*s_{NxN}     foreach e,k
    mxm_gpu2(u1,npts,*n,  dt,n2,*n,    u1s,npts,*n,  *nelts,13, 0);
    mxm_gpu2(u2,npts,*n,  dt,n2,*n,    u2s,npts,*n,  *nelts,13, 0);
    mxm_gpu2(u3,npts,*n,  dt,n2,*n,    u3s,npts,*n,  *nelts,13, 0);
  
    //       u*_{N^2xN} * dt_{NxN}  =  u*t_{N^2xN}   foreach e
    mxm_gpu2(u1,npts,n2,  dt,n2,*n,    u1t,npts,*n,  *nelts,5, 0);
    mxm_gpu2(u2,npts,n2,  dt,n2,*n,    u2t,npts,*n,  *nelts,5, 0);
    mxm_gpu2(u3,npts,n2,  dt,n2,*n,    u3t,npts,*n,  *nelts,5, 0);
  } else {
    // todo: fork threads or do async launches
    //       d_{NxN}   *  u*_{NxN^2} = u*r_{NxN^2}   foreach e
    mxm_gpu2(d,n2,*n,     u1,npts,*n,  u1r,npts,n2,  *nelts,6, devid);
    mxm_gpu2(d,n2,*n,     u2,npts,*n,  u2r,npts,n2,  *nelts,6, devid);
    mxm_gpu2(d,n2,*n,     u3,npts,*n,  u3r,npts,n2,  *nelts,6, devid);
  
    //       u*_{NxN}  *  dt_{NxN}  =  u*s_{NxN}     foreach e,k
    mxm_gpu2(u1,npts,*n,  dt,n2,*n,    u1s,npts,*n,  *nelts,13, devid);
    mxm_gpu2(u2,npts,*n,  dt,n2,*n,    u2s,npts,*n,  *nelts,13, devid);
    mxm_gpu2(u3,npts,*n,  dt,n2,*n,    u3s,npts,*n,  *nelts,13, devid);
  
    //       u*_{N^2xN} * dt_{NxN}  =  u*t_{N^2xN}   foreach e
    mxm_gpu2(u1,npts,n2,  dt,n2,*n,    u1t,npts,*n,  *nelts,5, devid);
    mxm_gpu2(u2,npts,n2,  dt,n2,*n,    u2t,npts,*n,  *nelts,5, devid);
    mxm_gpu2(u3,npts,n2,  dt,n2,*n,    u3t,npts,*n,  *nelts,5, devid);
  }
}

// Sets up the curl kernel
void curl_gpu_(double* u1r, double* u1s, double* u1t,
               double* u2r, double* u2s, double* u2t,
               double* u3r, double* u3s, double* u3t,
               double* rxmn,double* sxmn,double* txmn,
               double* rymn,double* symn,double* tymn,
               double* rzmn,double* szmn,double* tzmn,
               double* w1,  double* w2,  double* w3, double* w3m, int* nxyz, int* nelts){
  /*device variables*/
  double *dev_rxmn, *dev_rymn, *dev_rzmn
        ,*dev_sxmn, *dev_symn, *dev_szmn
        ,*dev_txmn, *dev_tymn, *dev_tzmn
        ,*dev_u1r, *dev_u1s, *dev_u1t
        ,*dev_u2r, *dev_u2s, *dev_u2t
        ,*dev_u3r, *dev_u3s, *dev_u3t
        ,*dev_w1, *dev_w2, *dev_w3, *dev_w3m;
  int nptsz=*nxyz*(*nelts)*sizeof(double);
  /*malloc and memcopy H2D*/
  hipMalloc(&dev_rxmn,nptsz);
  hipMalloc(&dev_rymn,nptsz);
  hipMalloc(&dev_rzmn,nptsz);
  hipMalloc(&dev_sxmn,nptsz);
  hipMalloc(&dev_symn,nptsz);
  hipMalloc(&dev_szmn,nptsz);
  hipMalloc(&dev_txmn,nptsz);
  hipMalloc(&dev_tymn,nptsz);
  hipMalloc(&dev_tzmn,nptsz);
  hipMalloc(&dev_u1r, nptsz);
  hipMalloc(&dev_u1s, nptsz);
  hipMalloc(&dev_u1t, nptsz);
  hipMalloc(&dev_u2r, nptsz);
  hipMalloc(&dev_u2s, nptsz);
  hipMalloc(&dev_u2t, nptsz);
  hipMalloc(&dev_u3r, nptsz);
  hipMalloc(&dev_u3s, nptsz);
  hipMalloc(&dev_u3t, nptsz);
  hipMalloc(&dev_w3m, nptsz);
  hipMalloc(&dev_w1,  nptsz);
  hipMalloc(&dev_w2,  nptsz);
  hipMalloc(&dev_w3,  nptsz);
  hipMemcpy(dev_rxmn,rxmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_rymn,rymn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_rzmn,rzmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_sxmn,sxmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_symn,symn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_szmn,szmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_txmn,txmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_tymn,tymn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_tzmn,tzmn,nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u1r, u1r, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u1s, u1s, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u1t, u1t, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u2r, u2r, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u2s, u2s, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u2t, u2t, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u3r, u3r, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u3s, u3s, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_u3t, u3t, nptsz,hipMemcpyHostToDevice);
  hipMemcpy(dev_w3m, w3m, nptsz,hipMemcpyHostToDevice);
  /*thread grid dimensions*/
  dim3 dimBlock, dimGrid;
  dimBlock.x=*nxyz; dimGrid.x=(15+dimBlock.x-1)/dimBlock.x;
  curl_vanilla<<<dimGrid,dimBlock>>>(
    dev_rxmn,dev_rymn,dev_rzmn,
    dev_sxmn,dev_symn,dev_szmn,
    dev_txmn,dev_tymn,dev_tzmn,
    dev_u1r,dev_u1s,dev_u1t,
    dev_u2r,dev_u2s,dev_u2t,
    dev_u3r,dev_u3s,dev_u3t,
    dev_w3m,*nxyz,*nelts,
    dev_w1, dev_w2, dev_w3
  );
  hipMemcpy(w1,dev_w1,nptsz,hipMemcpyDeviceToHost);
  hipMemcpy(w2,dev_w2,nptsz,hipMemcpyDeviceToHost);
  hipMemcpy(w3,dev_w3,nptsz,hipMemcpyDeviceToHost);
  hipFree(dev_rxmn);
  hipFree(dev_rymn);
  hipFree(dev_rzmn);
  hipFree(dev_sxmn);
  hipFree(dev_symn);
  hipFree(dev_szmn);
  hipFree(dev_txmn);
  hipFree(dev_tymn);
  hipFree(dev_tzmn);
  hipFree(dev_u1r);
  hipFree(dev_u1s);
  hipFree(dev_u1t);
  hipFree(dev_u2r);
  hipFree(dev_u2s);
  hipFree(dev_u2t);
  hipFree(dev_u3r);
  hipFree(dev_u3s);
  hipFree(dev_u3t);
  hipFree(dev_w3m);
  hipFree(dev_w1);
  hipFree(dev_w2);
  hipFree(dev_w3);
  hipDeviceSynchronize();
}

