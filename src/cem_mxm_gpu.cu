/*
 * mxm_gpu.cu
 *  @author azamat, mmin
 *  @since  July 13, 2012
 */

#include <stdio.h>
#include <hip/hip_runtime.h>


#define KERNEL  1
#define TILE   8 //autotune-able
#define VERBOSE 1
#define CUBLAS 0

#if VERBOSE
int dbg=1;
#else
int dbg=0;
#endif

#if CUBLAS
#include <cublas_v2.h>
cublasHandle_t cublas_h;
cublasStatus_t stat = cublasCreate(&cublas_h);
#endif

static int once=0;
hipEvent_t tstart, tstop, start, stop;
float kern=0.0f, xfer=0.0f;

int mpirank=0, devid=0;

#define onceMallocMemcpy(x,dbg) do{                                \
  if ((x)->sync&0x1) {                                             \
    hipMalloc(&(x)->dev,(x)->sz);                                 \
    if(dbg){                                                       \
      printf("r%d.d%d cudaMalloc'ed:     %s, %d B\n",mpirank,devid,(x)->vname,(x)->sz);  \
    }                                                              \
    (x)->sync^=0x1;                                                \
  }                                                                \
  if ((x)->sync&0x2) {                                             \
    hipEventRecord(tstart,0);                                     \
    hipMemcpy((x)->dev,(x)->host,(x)->sz,hipMemcpyHostToDevice); \
    hipEventRecord(tstop,0); hipEventSynchronize(tstop);         \
    if(dbg){                                                       \
      hipEventElapsedTime(&xfer,tstart,tstop);                    \
      printf("r%d.d%d cudaMemcpy'ed H2D: %s, %d B, %f ms, %.2f MB/s\n",mpirank,devid,(x)->vname,(x)->sz,xfer,(1e3f*(x)->sz)/(xfer*(1<<20)));  \
    }                                                              \
    (x)->sync^=0x2;                                                \
  }                                                                \
}while(0)
#define onceMemcpyFree(x,dbg) do{                                  \
  if ((x)->sync&0x4) {                                             \
    hipEventRecord(tstart,0);                                     \
    hipMemcpy((x)->host,(x)->dev,(x)->sz,hipMemcpyDeviceToHost); \
    hipEventRecord(tstop,0); hipEventSynchronize(tstop);         \
    if(dbg){                                                       \
      hipEventElapsedTime(&xfer,tstart,tstop);                    \
      printf("r%d.d%d cudaMemcpy'ed D2H: %s, %d B, %f ms, %.2f MB/s\n",mpirank,devid,(x)->vname,(x)->sz,xfer,(1e3f*(x)->sz)/(xfer*(1<<20)));  \
    }                                                              \
    (x)->sync^=0x4;                                                \
  }                                                                \
  if ((x)->sync&0x8) {                                             \
    hipFree((x)->dev);                                            \
    if(dbg){                                                       \
      printf("r%d.d%d cudaFree'ed:       %s\n",mpirank,devid,(x)->vname);                \
    }                                                              \
    (x)->sync^=0x8;                                                \
  }                                                                \
}while(0)


//=============================================================================
extern "C" {
  struct memptr {
    int sync; //sync flags: 0x1->allocate, 0x2->copy H2D, 0x4->copy D2H, 0x8->deallocate
    int sz;
    double* host;
    double* dev;
    char* vname;
  };
  typedef struct memptr memptr_t;
  void mxm_std_gpu_(double* a, int* m, double* b, int* n, double* c, int* p);
  void local_grad3_gpu_(
    memptr_t *u1r, memptr_t *u1s, memptr_t *u1t,
    memptr_t *u2r, memptr_t *u2s, memptr_t *u2t,
    memptr_t *u3r, memptr_t *u3s, memptr_t *u3t,
    memptr_t *u1 , memptr_t *u2 , memptr_t *u3 ,
    memptr_t *mp_d, memptr_t *mp_dt,
    int *n, int *nelts, int *lpts1, int *rank);
  void curl_gpu_(
    memptr_t *u1r, memptr_t *u1s, memptr_t *u1t,
    memptr_t *u2r, memptr_t *u2s, memptr_t *u2t,
    memptr_t *u3r, memptr_t *u3s, memptr_t *u3t,
    memptr_t *rxmn,memptr_t *sxmn,memptr_t *txmn,
    memptr_t *rymn,memptr_t *symn,memptr_t *tymn,
    memptr_t *rzmn,memptr_t *szmn,memptr_t *tzmn,
    memptr_t *w1,  memptr_t *w2,  memptr_t *w3,
    memptr_t *w3mn,
    int *nxyz, int *nelts, int *lpts1);
}


//=============================================================================
// basic curl kernel impl
// this source: 44 registers/thread, 200 bytes cmem[0]
// can improve bandwidth: ~83% of peak (140 GB/s) due gmem cache misses; global_load_miss/inst_issued=26%
__global__ void curl_k(
    const double* __restrict__ rxmn,const double* __restrict__ rymn,const double* __restrict__ rzmn,
    const double* __restrict__ sxmn,const double* __restrict__ symn,const double* __restrict__ szmn,
    const double* __restrict__ txmn,const double* __restrict__ tymn,const double* __restrict__ tzmn,
    const double* __restrict__ u1r, const double* __restrict__ u1s, const double* __restrict__ u1t,
    const double* __restrict__ u2r, const double* __restrict__ u2s, const double* __restrict__ u2t,
    const double* __restrict__ u3r, const double* __restrict__ u3s, const double* __restrict__ u3t,
    const double* __restrict__ w3mn,const int lpts1, 
    double* const __restrict__ w1
  ){
  const int k=blockIdx.x*blockDim.x+threadIdx.x;
  const double w3mk=w3mn[threadIdx.x];
  double* const __restrict__ w2 = &w1[lpts1];
  double* __restrict__ const w3 = &w2[lpts1];

  w1[k]= w3mk*u3r[k]*rymn[k]
       + w3mk*u3s[k]*symn[k]
       + w3mk*u3t[k]*tymn[k]
       - w3mk*u2r[k]*rzmn[k]
       - w3mk*u2s[k]*szmn[k]
       - w3mk*u2t[k]*tzmn[k];

  w2[k]= w3mk*u1r[k]*rzmn[k]
       + w3mk*u1s[k]*szmn[k]
       + w3mk*u1t[k]*tzmn[k]
       - w3mk*u3r[k]*rxmn[k]
       - w3mk*u3s[k]*sxmn[k]
       - w3mk*u3t[k]*txmn[k];

  w3[k]= w3mk*u2r[k]*rxmn[k]
       + w3mk*u2s[k]*sxmn[k]
       + w3mk*u2t[k]*txmn[k]
       - w3mk*u1r[k]*rymn[k]
       - w3mk*u1s[k]*symn[k]
       - w3mk*u1t[k]*tymn[k];
}


//=============================================================================
// basic multi-mxm impl
__global__ void mxm_vanilla(const double* __restrict__ a, const int m,
                            const double* __restrict__ b, const int n,
                            double* __restrict__ c, const int p,
                            const int nelts, const int ldims){
  const int row=blockIdx.y*blockDim.y+threadIdx.y;
  const int col=blockIdx.x*blockDim.x+threadIdx.x;
  if(row<m && col<p){ //eliminate out-of-bounds threads
    double s;
    int lda=( ldims&0x1)    *m*n    //if a's bit (0x1) is set, its leading dim is of size m*n 
      , ldb=((ldims&0x2)>>1)*n*p
      , ldc=((ldims&0x4)>>2)*m*p
      , ldi=((ldims&0x8)>>3)*m*n*p; //for inner dimensions
    if(ldims<8){ //no inner iterations
      for(int e=0; e<nelts; e++){
        s=0.0;
        for(int k=0; k<n; k++){
          s+=a[e*lda+k*m+row]*b[e*ldb+col*n+k];
        }
        c[e*ldc+col*m+row]=s;
      }
    }else{
      for(int e=0; e<nelts; e++){
        for(int i=0; i<m; i++){
          s=0.0;
          for(int k=0; k<n; k++){
            s+=a[e*ldi+i*lda+k*m+row]*b[col*n+k];
          }
          c[e*ldi+i*ldc+col*m+row]=s;
        }
      }
    }
  }
}


//=============================================================================
// mxm: R = D * U
__global__ void mxmr_any(
    const double* __restrict__ a, const int m,
    const double* __restrict__ b, const int n,
          double* __restrict__ c, const int p){
  const int col=threadIdx.z*blockDim.y+threadIdx.y;
  register double s=0.0;
  #pragma unroll
  for(int k=0; k<n; k++){
    s+=a[threadIdx.x+m*k]*b[blockIdx.x*n*p+col*n+k];
  }
  c[blockIdx.x*m*p+col*m+threadIdx.x]=s;
}

// bandwidth: ~44% of peak (75 GB/s) 
__global__ void mxmr8(
    const double* __restrict__ a,
    const double* __restrict__ b,
          double* __restrict__ c){
  __shared__ double as[64], bs[512];
  const int col=8*threadIdx.z+threadIdx.y;
  as[8*threadIdx.y+threadIdx.x]=a[8*threadIdx.y+threadIdx.x];
  bs[8*col+threadIdx.x]=b[512*blockIdx.x+8*col+threadIdx.x];
  __syncthreads();
  register double s=0.0;
  #pragma unroll 8
  for(int k=0; k<8; k++){
    s+=as[8*k+threadIdx.x]*bs[8*col+k];
  }
  c[512*blockIdx.x+8*col+threadIdx.x]=s;
}


//=============================================================================
// mxm: S = U * D'
__global__ void mxms_any(
    const double* __restrict__ a, const int m,
    const double* __restrict__ b, const int n,
          double* __restrict__ c, const int p){
  const int col=threadIdx.z*blockDim.y+threadIdx.y;
  register double s=0.0;
  #pragma unroll
  for(int k=0; k<n; k++){
    s+=a[blockIdx.x*m*n*p+threadIdx.z*m*n+m*k+threadIdx.x]*b[threadIdx.y*n+k];
  }
  c[blockIdx.x*m*n*p+col*m+threadIdx.x]=s;
}

// bandwidth: ~44% of peak (74 GB/s) 
__global__ void mxms8(
    const double* __restrict__ a,
    const double* __restrict__ b,
          double* __restrict__ c){
  __shared__ double as[512], bs[64];
  const int col=8*threadIdx.z+threadIdx.y;
  as[8*col+threadIdx.x]=a[512*blockIdx.x+8*col+threadIdx.x];
  bs[8*threadIdx.y+threadIdx.x]=b[8*threadIdx.y+threadIdx.x];
  __syncthreads();
  register double s=0.0;
  #pragma unroll 8
  for(int k=0; k<8; k++){
    s+=as[64*threadIdx.z+8*k+threadIdx.x]*bs[8*threadIdx.y+k];
  }
  c[512*blockIdx.x+8*col+threadIdx.x]=s;
}


//=============================================================================
// mxm: T = U * D'
__global__ void mxmt_any(
    const double* __restrict__ a, const int m,
    const double* __restrict__ b, const int n,
          double* __restrict__ c, const int p){
  const int row=threadIdx.z*blockDim.x+threadIdx.x;
  register double s=0.0;
  #pragma unroll
  for(int k=0; k<n; k++){
    s+=a[blockIdx.x*m*n+row+k*m]*b[threadIdx.y*n+k];
  }
  c[blockIdx.x*m*p+threadIdx.y*m+row]=s;
}

__global__ void mxmt8(
    const double* __restrict__ a,
    const double* __restrict__ b,
          double* __restrict__ c){
  __shared__ double as[512], bs[64];
  const int row=8*threadIdx.z+threadIdx.x;
  as[64*threadIdx.y+row]=a[512*blockIdx.x+64*threadIdx.y+row];
  bs[ 8*threadIdx.y+threadIdx.x]=b[8*threadIdx.y+threadIdx.x];
  __syncthreads();
  register double s=0.0;
  #pragma unroll 8
  for(int k=0; k<8; k++){
    s+=as[8*threadIdx.z+threadIdx.x+k*64]*bs[8*threadIdx.y+k];
  }
  c[512*blockIdx.x+64*threadIdx.y+row]=s;
}


//=============================================================================
// mxm with 1D arrays
__global__ void mxm_1d(double* a, const int m, double* b, const int n, double* c, const int p){
  const int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i<m){
    for(int k=0; k<p; k++){
      double s=0.0;
      for(int j=0; j<n; j++){
        s+=a[j*m+i]*b[k*n+j];
      }
      c[k*m+i]=s;
    }
  }
}


// mxm with 2D arrays
__global__ void mxm_shared(double* a, const int m, double* b, const int n, double* c, const int p){
  __shared__ double as[TILE][TILE];
  __shared__ double bs[TILE][TILE];
  int bx=blockIdx.x, by=blockIdx.y, tx=threadIdx.x, ty=threadIdx.y;
  const int row=by*TILE+ty;
  const int col=bx*TILE+tx;
  double s=0.0;
  for(int t=0;t<m/TILE;t++){
    as[ty][tx]=a[col*m+t*TILE+tx];
    bs[ty][tx]=b[col*n+t*TILE+ty];
    __syncthreads();
    for(int k=0; k<TILE; k++){
      s+=as[ty][k]*bs[k][tx];
    }
    __syncthreads();
    c[col*m+row]=s;
  }
}


// globally-visible basic mxm implementation for small matrices
void mxm_std_gpu_(double* a, int* m, double* b, int* n, double* c, int* p){
  /*device variables*/
  double *dev_a, *dev_b, *dev_c;
  int sizeofA=*m*(*n)*sizeof(double)
    , sizeofB=*n*(*p)*sizeof(double)
    , sizeofC=*m*(*p)*sizeof(double);
  /*malloc and memcopy data H2D*/
  hipMalloc(&dev_a,sizeofA);
  hipMalloc(&dev_b,sizeofB);
  hipMalloc(&dev_c,sizeofC);
  hipMemcpy(dev_a,a,sizeofA,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,sizeofB,hipMemcpyHostToDevice);
  /*thread dimensions*/
  dim3 dimBlock, dimGrid;
#if KERNEL==1
  dimBlock.x=TILE; dimGrid.x=(*p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(*m+dimBlock.y-1)/dimBlock.y;
  mxm_vanilla<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p,1,0);
#elif KERNEL==2
  dimBlock.x=TILE; dimGrid.x=(*m+dimBlock.x-1)/dimBlock.x;
  mxm_1d<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p);
#else
  dimBlock.x=TILE; dimGrid.x=(*p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(*m+dimBlock.y-1)/dimBlock.y;
  mxm_shared<<<dimGrid,dimBlock>>>(dev_a,*m,dev_b,*n,dev_c,*p);
#endif
  /*memcopy D2H*/
  hipMemcpy(c,dev_c,sizeofC,hipMemcpyDeviceToHost);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}


// sets up the aggregated mxm kernel launch
void mxm_gpu2(double* a, int as, int m
             ,double* b, int bs, int n
             ,double* c, int cs, int p
             ,int nelts, int mask, int dev){
  hipSetDevice(dev);
  /*device variables*/
  double *dev_a, *dev_b, *dev_c;
  int sizeofA=as*sizeof(double)
    , sizeofB=bs*sizeof(double)
    , sizeofC=cs*sizeof(double);
  /*malloc and memcopy H2D*/
  hipMalloc(&dev_a,sizeofA);
  hipMalloc(&dev_b,sizeofB);
  hipMalloc(&dev_c,sizeofC);
  hipMemcpy(dev_a,a,sizeofA,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,sizeofB,hipMemcpyHostToDevice);
  /*thread grid dimensions*/
  dim3 dimBlock, dimGrid;
  dimBlock.x=TILE; dimGrid.x=(p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(m+dimBlock.y-1)/dimBlock.y;
  mxm_vanilla<<<dimGrid,dimBlock>>>(dev_a,m, dev_b,n, dev_c,p, nelts,mask);
  /*memcopy D2H*/
  hipMemcpy(c,dev_c,sizeofC,hipMemcpyDeviceToHost);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}

//=============================================================================
// sets up the aggregated mxm kernel launch
void mxm_gpu_agg(memptr_t *a, int m
                ,memptr_t *b, int n
                ,memptr_t *c, int p
                ,int nelts, int mask, int dev){
  hipSetDevice(dev);
  /*malloc and memcopy H2D*/
  onceMallocMemcpy(a,dbg);
  onceMallocMemcpy(b,dbg);
  onceMallocMemcpy(c,dbg);
  /*thread grid dimensions*/
  dim3 dimBlock, dimGrid;
  dimBlock.x=TILE; dimGrid.x=(p+dimBlock.x-1)/dimBlock.x;
  dimBlock.y=TILE; dimGrid.y=(m+dimBlock.y-1)/dimBlock.y;
  mxm_vanilla<<<dimGrid,dimBlock>>>(a->dev,m, b->dev,n, c->dev,p, nelts,mask);
  /*memcopy D2H and dealloc*/
  onceMemcpyFree(a,dbg);
  onceMemcpyFree(b,dbg);
  onceMemcpyFree(c,dbg);
}


//=============================================================================
/**
 * Performs aggregated mxm for all elements at once.
 *
 * foreach e in 0..nelts
 *   u@r_{NxN^2} = d_{NxN} * u@_{NxN^2}^{e} // here @ is either 1, 2 or 3
 *   foreach k in 0..N
 *     u@s_{NxN}^{k} = u@_{NxN}^{k,e} * dt_{NxN}
 *   u@t_{N^2xN} = u@_{N^2xN}^{e} * dt_{NxN}
 */
void local_grad3_gpu_(memptr_t *u1r, memptr_t *u1s, memptr_t *u1t,  
                      memptr_t *u2r, memptr_t *u2s, memptr_t *u2t,  
                      memptr_t *u3r, memptr_t *u3s, memptr_t *u3t,  
                      memptr_t *u1 , memptr_t *u2 , memptr_t *u3 ,  
                      memptr_t *d,   memptr_t *dt,
                      int *n, int *nelts, int *lpts1, int *rank){
  int n1=*n, n2=n1*n1, n3=n1*n2, ne=*nelts;
  float gbytes = 1e3f*((2*ne*n3+n2)*3*8.0f)/(1<<30);
  float gflops = 1e3f*2*n3*n1*ne*3/(1<<30);

  // select the device
  int devs = 0;
  hipGetDeviceCount(&devs);
  if (devs==1) {
    devid = 0;
  } else {
    devid = *rank%2;
  }
  hipSetDevice(devid);
  mpirank=*rank;

  if (!once) {
    d->vname   = "d";
    dt->vname  = "dt";
    u1r->vname = "u1r";
    u1s->vname = "u1s";
    u1t->vname = "u1t";
    u2r->vname = "u2r";
    u2s->vname = "u2s";
    u2t->vname = "u2t";
    u3r->vname = "u3r";
    u3s->vname = "u3s";
    u3t->vname = "u3t";
    u1->vname  = "u1";
    u2->vname  = "u2";
    u3->vname  = "u3";
    hipEventCreate(&tstart); hipEventCreate(&tstop);
    hipEventCreate(&start);  hipEventCreate(&stop);
  }

  onceMallocMemcpy(d,  dbg);
  onceMallocMemcpy(u1r,dbg);
  onceMallocMemcpy(u2r,dbg);
  onceMallocMemcpy(u3r,dbg);
  // u1,u2,u3 are contiguous, do a single transfer
  u1->sz=*lpts1*3*sizeof(double);
  onceMallocMemcpy(u1, dbg);
  u2->dev=u1->dev+(*lpts1);
  u3->dev=u2->dev+(*lpts1);

  /*thread grid dimensions*/
  dim3 dimBlock, dimGrid;

  hipEventRecord(start,0);

#if CUBLAS
  const double alpha = 1.0;
  const double beta  = 0.0;
  int inci, incj;
  for(int i=0; i<ne; i++){
    inci = i*n3;
    for(int j=0; j<*n; j++){
      incj = j*n2;
      cublasDgemm(cublas_h, CUBLAS_OP_N, CUBLAS_OP_N, *n,*n,*n, &alpha,
        d->dev,*n, u1->dev+inci+incj,*n, &beta, u1r->dev+inci+incj,*n);

      cublasDgemm(cublas_h, CUBLAS_OP_N, CUBLAS_OP_N, *n,*n,*n, &alpha,
        d->dev,*n, u2->dev+inci+incj,*n, &beta, u2r->dev+inci+incj,*n);

      cublasDgemm(cublas_h, CUBLAS_OP_N, CUBLAS_OP_N, *n,*n,*n, &alpha,
        d->dev,*n, u3->dev+inci+incj,*n, &beta, u3r->dev+inci+incj,*n);
    }
  } // this gets 0.19 GB/s
#else
  /* D_{NxN} * U_{NxN^2} = R_{NxN^2} foreach e */
  dimBlock.x=*n; dimBlock.y=*n, dimBlock.z=*n;
  dimGrid.x=ne;  dimGrid.y=1;   dimGrid.z=1;
  if (*n==8){
    mxmr8<<<dimGrid,dimBlock>>>(d->dev, u1->dev, u1r->dev);
    mxmr8<<<dimGrid,dimBlock>>>(d->dev, u2->dev, u2r->dev);
    mxmr8<<<dimGrid,dimBlock>>>(d->dev, u3->dev, u3r->dev);
  }else{
    // todo: dispatch to other specialized mxmr kernels
    mxmr_any<<<dimGrid,dimBlock>>>(d->dev,*n, u1->dev,*n, u1r->dev,n2);
    mxmr_any<<<dimGrid,dimBlock>>>(d->dev,*n, u2->dev,*n, u2r->dev,n2);
    mxmr_any<<<dimGrid,dimBlock>>>(d->dev,*n, u3->dev,*n, u3r->dev,n2);
  }
#endif
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&kern,start,stop);
  if(dbg){
    printf("r%d.d%d r kernel time:     %f ms, eff.bw: %f GB/s, perf: %f GFlop/s\n",mpirank,devid,kern,gbytes/kern,gflops/kern);
  }

  onceMallocMemcpy(dt, dbg);
  onceMallocMemcpy(u1s,dbg);
  onceMallocMemcpy(u2s,dbg);
  onceMallocMemcpy(u3s,dbg);
  /* U_{NxN} * D'_{NxN} = S_{NxN} foreach e,k */
  dimBlock.x=*n; dimBlock.y=*n; dimBlock.z=*n;
  dimGrid.x=ne;  dimGrid.y=1;   dimGrid.z=1;
  hipEventRecord(start,0);
  if (*n==8){
    mxms8<<<dimGrid,dimBlock>>>(u1->dev, dt->dev, u1s->dev);
    mxms8<<<dimGrid,dimBlock>>>(u2->dev, dt->dev, u2s->dev);
    mxms8<<<dimGrid,dimBlock>>>(u3->dev, dt->dev, u3s->dev);
  }else{
    // todo: dispatch to other specialized mxms kernels
    mxms_any<<<dimGrid,dimBlock>>>(u1->dev,*n, dt->dev,*n, u1s->dev,*n);
    mxms_any<<<dimGrid,dimBlock>>>(u2->dev,*n, dt->dev,*n, u2s->dev,*n);
    mxms_any<<<dimGrid,dimBlock>>>(u3->dev,*n, dt->dev,*n, u3s->dev,*n);
  }
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&kern,start,stop);
  if(dbg){
    printf("r%d.d%d s kernel time:     %f ms, eff.bw: %f GB/s, perf: %f GFlop/s\n",mpirank,devid,kern,gbytes/kern,gflops/kern);
  }

  onceMallocMemcpy(u1t,dbg);
  onceMallocMemcpy(u2t,dbg);
  onceMallocMemcpy(u3t,dbg);
  /* U_{N^2xN} * D'_{NxN} = T_{N^2xN} foreach e */
  dimBlock.x=*n; dimBlock.y=*n; dimBlock.z=*n;
  dimGrid.x=ne;  dimGrid.y=1;   dimGrid.z=1;
  hipEventRecord(start,0);
  if (*n==8){
    mxmt8<<<dimGrid,dimBlock>>>(u1->dev, dt->dev, u1t->dev);
    mxmt8<<<dimGrid,dimBlock>>>(u2->dev, dt->dev, u2t->dev);
    mxmt8<<<dimGrid,dimBlock>>>(u3->dev, dt->dev, u3t->dev);
  }else{
    // todo: dispatch to other specialized mxmt kernels
    mxmt_any<<<dimGrid,dimBlock>>>(u1->dev,n2, dt->dev,*n, u1t->dev,*n);
    mxmt_any<<<dimGrid,dimBlock>>>(u2->dev,n2, dt->dev,*n, u2t->dev,*n);
    mxmt_any<<<dimGrid,dimBlock>>>(u3->dev,n2, dt->dev,*n, u3t->dev,*n);
  }
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&kern,start,stop);
  if(dbg){
    printf("r%d.d%d t kernel time:     %f ms, eff.bw: %f GB/s, perf: %f GFlop/s\n",mpirank,devid,kern,gbytes/kern,gflops/kern);
  }

  // nothing to copy D2H or to free
  //cudaDeviceSynchronize();
}

//=============================================================================
// Sets up the curl kernel
void curl_gpu_(memptr_t *u1r,  memptr_t *u1s,  memptr_t *u1t,
               memptr_t *u2r,  memptr_t *u2s,  memptr_t *u2t,
               memptr_t *u3r,  memptr_t *u3s,  memptr_t *u3t,
               memptr_t *rxmn, memptr_t *sxmn, memptr_t *txmn,
               memptr_t *rymn, memptr_t *symn, memptr_t *tymn,
               memptr_t *rzmn, memptr_t *szmn, memptr_t *tzmn,
               memptr_t *w1,   memptr_t *w2,   memptr_t *w3,
               memptr_t *w3mn, int *nxyz, int *nelts, int *lpts1){
  int n3=*nxyz, npts=*nelts*n3;
  float gbytes = 1e3f*((n3+21*npts)*8.0f)/(1<<30);
  float gflops = 1e3f*(51*npts)/(1<<30);
  if (!once){
    rxmn->vname="rxmn"; sxmn->vname="sxmn"; txmn->vname="txmn";
    rymn->vname="rymn"; symn->vname="symn"; tymn->vname="tymn";
    rzmn->vname="rzmn"; szmn->vname="szmn"; tzmn->vname="tzmn";
    w3mn->vname="w3mn";
    w1->vname="w1"; w2->vname="w2"; w3->vname="w3";
    once=1;
  }
  /*malloc and memcopy H2D*/
  onceMallocMemcpy(rxmn,dbg);
  onceMallocMemcpy(rymn,dbg);
  onceMallocMemcpy(rzmn,dbg);
  onceMallocMemcpy(sxmn,dbg);
  onceMallocMemcpy(symn,dbg);
  onceMallocMemcpy(szmn,dbg);
  onceMallocMemcpy(txmn,dbg);
  onceMallocMemcpy(tymn,dbg);
  onceMallocMemcpy(tzmn,dbg);
  onceMallocMemcpy(w3mn,dbg);
  onceMallocMemcpy(u1r, dbg);
  onceMallocMemcpy(u1s, dbg);
  onceMallocMemcpy(u1t, dbg);
  onceMallocMemcpy(u2r, dbg);
  onceMallocMemcpy(u2s, dbg);
  onceMallocMemcpy(u2t, dbg);
  onceMallocMemcpy(u3r, dbg);
  onceMallocMemcpy(u3s, dbg);
  onceMallocMemcpy(u3t, dbg);
  // w1,w2,w3 are contiguous, do a single transfer
  w1->sz=*lpts1*3*sizeof(double);
  onceMallocMemcpy(w1,  dbg);
  /*thread grid dimensions*/
  dim3 dimBlock, dimGrid;
  dimBlock.x=*nxyz; dimGrid.x=*nelts;
  hipEventRecord(start,0);
  curl_k<<<dimGrid,dimBlock>>>(
    rxmn->dev,rymn->dev,rzmn->dev,
    sxmn->dev,symn->dev,szmn->dev,
    txmn->dev,tymn->dev,tzmn->dev,
    u1r->dev, u1s->dev, u1t->dev,
    u2r->dev, u2s->dev, u2t->dev,
    u3r->dev, u3s->dev, u3t->dev,
    w3mn->dev, *lpts1,
    w1->dev
  );
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&kern,start,stop);
  if(dbg){
    printf("r%d.d%d curl kernel time:  %f ms, eff.bw: %f GB/s, perf: %f GFlop/s\n",mpirank,devid,kern,gbytes/kern,gflops/kern);
  }
  onceMemcpyFree(rxmn,dbg);
  onceMemcpyFree(rymn,dbg);
  onceMemcpyFree(rzmn,dbg);
  onceMemcpyFree(sxmn,dbg);
  onceMemcpyFree(symn,dbg);
  onceMemcpyFree(szmn,dbg);
  onceMemcpyFree(txmn,dbg);
  onceMemcpyFree(tymn,dbg);
  onceMemcpyFree(tzmn,dbg);
  onceMemcpyFree(w3mn,dbg);
  onceMemcpyFree(u1r, dbg);
  onceMemcpyFree(u1s, dbg);
  onceMemcpyFree(u1t, dbg);
  onceMemcpyFree(u2r, dbg);
  onceMemcpyFree(u2s, dbg);
  onceMemcpyFree(u2t, dbg);
  onceMemcpyFree(u3r, dbg);
  onceMemcpyFree(u3s, dbg);
  onceMemcpyFree(u3t, dbg);
  onceMemcpyFree(w1,  dbg);
}

